#include "hip/hip_runtime.h"
#include "stdio.h"
#include "stdint.h"
#include "vector"
#include "linearprobing.h"

// 32 bit Murmur3 hash
__device__ uint32_t hash(uint32_t k, uint32_t mod)
{
    k ^= k >> 16;
    k *= 0x85ebca6b;
    k ^= k >> 13;
    k *= 0xc2b2ae35;
    k ^= k >> 16;
    return k & (mod-1);
}

// Create a hash table. For linear probing, this is just an array of KeyValues
HashTable create_hashtable(uint32_t capacity) 
{
    // Allocate memory
    KeyValue* hashtable;
    uint32_t* size;
    hipMalloc(&hashtable, sizeof(KeyValue) * capacity);
    hipMalloc(&size, sizeof(uint32_t));

    // Initialize hash table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(hashtable, 0xff, sizeof(KeyValue) * capacity);
    hipMemset(size, 0x0, sizeof(uint32_t));

    uint32_t size1;
    hipMemcpy(size, &size1, sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("    space used: %d\n", size1);

    return { hashtable, size, capacity };
}

// Insert the key/values in kvs into the hashtable
__global__ void gpu_hashtable_insert(HashTable ht, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < numkvs)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t value = kvs[threadid].value;
        uint32_t slot = hash(key, ht.capacity);

        while (true)
        {
            uint32_t prev = atomicCAS(&ht.hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty)
                atomicAdd(ht.size, 1); //new key space used
            if (prev == kEmpty || prev == key)
            {
                ht.hashtable[slot].value = value;
                return;
            }

            slot = (slot + 1) & (ht.capacity-1);
        }
    }
}
 
void insert_hashtable(HashTable& ht, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert<<<gridsize, threadblocksize>>>(ht, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU inserted %d items in %f ms (%f million keys/second)\n", 
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    uint32_t size;
    hipMemcpy(&size, ht.size, sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("    space used: %d\n", size);

    hipFree(device_kvs);
}

// Lookup keys in the hashtable, and return the values
__global__ void gpu_hashtable_lookup(HashTable ht, KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < ht.capacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key, ht.capacity);

        while (true)
        {
            if (ht.hashtable[slot].key == key)
            {
                kvs[threadid].value = ht.hashtable[slot].value;
                return;
            }
            if (ht.hashtable[slot].key == kEmpty)
            {
                kvs[threadid].value = kEmpty;
                return;
            }
            slot = (slot + 1) & (ht.capacity - 1);
        }
    }
}

void lookup_hashtable(HashTable& ht, KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_insert << <gridsize, threadblocksize >> > (ht, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU lookup %d items in %f ms (%f million keys/second)\n",
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Delete each key in kvs from the hash table, if the key exists
// A deleted key is left in the hash table, but its value is set to kEmpty
// Deleted keys are not reused; once a key is assigned a slot, it never moves
__global__ void gpu_hashtable_delete(HashTable ht, const KeyValue* kvs, unsigned int numkvs)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < ht.capacity)
    {
        uint32_t key = kvs[threadid].key;
        uint32_t slot = hash(key, ht.capacity);

        while (true)
        {
            if (ht.hashtable[slot].key == key)
            {
                ht.hashtable[slot].value = kEmpty;
                return;
            }
            if (ht.hashtable[slot].key == kEmpty)
            {
                return;
            }
            slot = (slot + 1) & (ht.capacity - 1);
        }
    }
}

void delete_hashtable(HashTable& ht, const KeyValue* kvs, uint32_t num_kvs)
{
    // Copy the keyvalues to the GPU
    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * num_kvs);
    hipMemcpy(device_kvs, kvs, sizeof(KeyValue) * num_kvs, hipMemcpyHostToDevice);

    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_insert, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = ((uint32_t)num_kvs + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_delete<< <gridsize, threadblocksize >> > (ht, device_kvs, (uint32_t)num_kvs);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    float seconds = milliseconds / 1000.0f;
    printf("    GPU delete %d items in %f ms (%f million keys/second)\n",
        num_kvs, milliseconds, num_kvs / (double)seconds / 1000000.0f);

    hipFree(device_kvs);
}

// Iterate over every item in the hashtable; return non-empty key/values
__global__ void gpu_iterate_hashtable(HashTable ht, KeyValue* kvs, uint32_t* kvs_size)
{
    unsigned int threadid = blockIdx.x * blockDim.x + threadIdx.x;
    if (threadid < ht.capacity) 
    {
        if (ht.hashtable[threadid].key != kEmpty) 
        {
            uint32_t value = ht.hashtable[threadid].value;
            if (value != kEmpty)
            {
                uint32_t size = atomicAdd(kvs_size, 1);
                kvs[size] = ht.hashtable[threadid];
            }
        }
    }
}

std::vector<KeyValue> iterate_hashtable(HashTable &ht)
{
    uint32_t* device_num_kvs;
    hipMalloc(&device_num_kvs, sizeof(uint32_t));
    hipMemset(device_num_kvs, 0, sizeof(uint32_t));

    KeyValue* device_kvs;
    hipMalloc(&device_kvs, sizeof(KeyValue) * ht.capacity/2);

    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_iterate_hashtable, 0, 0);

    int gridsize = (ht.capacity + threadblocksize - 1) / threadblocksize;
    gpu_iterate_hashtable<<<gridsize, threadblocksize>>>(ht, device_kvs, device_num_kvs);

    uint32_t num_kvs;
    hipMemcpy(&num_kvs, device_num_kvs, sizeof(uint32_t), hipMemcpyDeviceToHost);

    std::vector<KeyValue> kvs;
    kvs.resize(num_kvs);

    hipMemcpy(kvs.data(), device_kvs, sizeof(KeyValue) * num_kvs, hipMemcpyDeviceToHost);

    hipFree(device_kvs);
    hipFree(device_num_kvs);

    return kvs;
}

// Free the memory of the hashtable
void destroy_hashtable(HashTable &ht)
{
    hipFree(ht.hashtable);
    hipFree(ht.size);
}

// Move the original kv into the new one
__global__ void gpu_hashtable_move(HashTable ht, HashTable new_ht)
{
    unsigned int threadid = blockIdx.x*blockDim.x + threadIdx.x;
    if (threadid < ht.capacity)
    {
        uint32_t key = ht.hashtable[threadid].key;
        uint32_t value = ht.hashtable[threadid].value;
        uint32_t slot = hash(key, new_ht.capacity);
        if (key == kEmpty || value == kEmpty) return; //skip empty and deleted

        //copied from the basic insertion
        while (true)
        {
            uint32_t prev = atomicCAS(&new_ht.hashtable[slot].key, kEmpty, key);
            if (prev == kEmpty)
                atomicAdd(new_ht.size, 1); //new key space used
            if (prev == kEmpty || prev == key)
            {
                new_ht.hashtable[slot].value = value;
                return;
            }

            slot = (slot + 1) & (new_ht.capacity-1);
        }
    }
}
 
void resize_hashtable(HashTable& ht, uint32_t resize_k)
{
    HashTable new_ht = { nullptr, nullptr, ht.capacity * resize_k }

    // Allocate mem for the new table
    hipMalloc(&new_ht.hashtable, sizeof(KeyValue) * new_ht.capacity);
    hipMalloc(&new_ht.size, sizeof(uint32_t));

    // Initialize new table to empty
    static_assert(kEmpty == 0xffffffff, "memset expected kEmpty=0xffffffff");
    hipMemset(new_ht.hashtable, 0xff, sizeof(KeyValue) * new_ht.capacity);
    hipMemset(new_ht.size, 0x0, sizeof(uint32_t));

    // Now copy NON-empty keys and their values
    // Have CUDA calculate the thread block size
    int mingridsize;
    int threadblocksize;
    hipOccupancyMaxPotentialBlockSize(&mingridsize, &threadblocksize, gpu_hashtable_resize, 0, 0);

    // Create events for GPU timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Insert all the keys into the hash table
    int gridsize = (ht.capacity + threadblocksize - 1) / threadblocksize;
    gpu_hashtable_move<<<gridsize, threadblocksize>>>(ht, new_ht);

    hipEventRecord(stop);

    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("    GPU moved %d items in %f ms\n", 
        *new_ht.size, milliseconds);

    uint32_t size;
    hipMemcpy(&size, new_ht.size, sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("    space used: %d\n", size);

    //nuke the old table and reassign it to the new one
    hipFree(ht.hashtable);
    hipFree(ht.size);
    ht = new_ht;
}